#include "hip/hip_runtime.h"
#include <wb.h> //@@ wb include opencl.h for you
#include <math.h>
//@@ OpenCL Kernel
 
// OpenCL kernel. Each work item takes care of one element of c
const char *kernelSource =                                       "\n" \
"#pragma OPENCL EXTENSION cl_khr_fp64 : enable                    \n" \
"__kernel void vecAdd(  __global float *a,                       \n" \
"                       __global float *b,                       \n" \
"                       __global float *c,                       \n" \
"                       const unsigned int n)                    \n" \
"{                                                               \n" \
"    //Get our global thread ID                                  \n" \
"    int id = get_global_id(0);                                  \n" \
"                                                                \n" \
"    //Make sure we do not go out of bounds                      \n" \
"    if (id < n)                                                 \n" \
"        c[id] = a[id] + b[id];                                  \n" \
"}                                                               \n" \
                                                                "\n" ;

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;
	
	 cl_platform_id cpPlatform;        // OpenCL platform
    cl_device_id device_id;           // device ID
    cl_context context;               // context
    cl_command_queue queue;           // command queue
    cl_program program;               // program
    cl_kernel kernel;                 // kernel
	cl_event event=NULL;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 = ( float * )wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 = ( float * )wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = ( float * )malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  int n=inputLength;
  cl_int err;
 
    // Number of work items in each local work group
   size_t localSize = 64;
 
    // Number of total work items - localSize must be devisor
   size_t  globalSize = ceil(inputLength/(float)localSize)*localSize;
 
    // Bind to platform
    err = clGetPlatformIDs(1, &cpPlatform, NULL);
 
    // Get ID for the device
    err = clGetDeviceIDs(cpPlatform, CL_DEVICE_TYPE_GPU, 1, &device_id, NULL);
 
    // Create a context  
    context = clCreateContext(0, 1, &device_id, NULL, NULL, &err);
 
    // Create a command queue 
    queue = clCreateCommandQueue(context, device_id, 0, &err);
	
	 // Create the compute program from the source buffer
    program = clCreateProgramWithSource(context, 1,
                            (const char **) & kernelSource, NULL, &err);
 
    // Build the program executable 
    clBuildProgram(program, 0, NULL, NULL, NULL, NULL);
 
    // Create the compute kernel in the program we wish to run
    kernel = clCreateKernel(program, "vecAdd", &err);
  int size = inputLength* sizeof(float);
  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
 // Device input buffers
  cl_mem d_a;
  cl_mem d_b;
 // Device output buffer
  cl_mem d_c;	
	
  d_a = clCreateBuffer(context, CL_MEM_READ_ONLY, size, NULL, NULL);
  d_b = clCreateBuffer(context, CL_MEM_READ_ONLY, size, NULL, NULL);
  d_c = clCreateBuffer(context, CL_MEM_WRITE_ONLY, size, NULL, NULL);
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  err = clEnqueueWriteBuffer(queue, d_a, CL_TRUE, 0,
                                   size, hostInput1, 0, NULL, NULL);
  wbTime_stop(GPU, "Copying input memory to the GPU.");
  err |= clEnqueueWriteBuffer(queue, d_b, CL_TRUE, 0,
                                   size,hostInput2, 0, NULL, NULL);
  //@@ Initialize the grid and block dimensions here

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here

 // Set the arguments to our compute kernel
    err  = clSetKernelArg(kernel, 0, sizeof(cl_mem), &d_a);
    err |= clSetKernelArg(kernel, 1, sizeof(cl_mem), &d_b);
    err |= clSetKernelArg(kernel, 2, sizeof(cl_mem), &d_c);
    err |= clSetKernelArg(kernel, 3, sizeof(unsigned int), &n);
 
    // Execute the kernel over the entire range of the data set  
    err = clEnqueueNDRangeKernel(queue, kernel, 1, NULL, &globalSize, &localSize,0, NULL, &event);
 
    // Wait for the command queue to get serviced before reading back results
    clFinish(queue);
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  clEnqueueReadBuffer(queue, d_c, CL_TRUE, 0,
                                size, hostOutput, 0, NULL, NULL );
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  clReleaseMemObject(d_a);
  clReleaseMemObject(d_b);
  clReleaseMemObject(d_c);
		clReleaseProgram(program);
    clReleaseKernel(kernel);
    clReleaseCommandQueue(queue);
    clReleaseContext(context);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
	


  return 0;
}
